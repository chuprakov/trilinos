#include "hip/hip_runtime.h"
/** \HEADER
 *************************************************************************
 *
 *                            Kokkos
 *                 Copyright 2010 Sandia Corporation
 *
 *  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
 *  the U.S. Government retains certain rights in this software.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are
 *  met:
 *
 *  1. Redistributions of source code must retain the above copyright
 *  notice, this list of conditions and the following disclaimer.
 *
 *  2. Redistributions in binary form must reproduce the above copyright
 *  notice, this list of conditions and the following disclaimer in the
 *  documentation and/or other materials provided with the distribution.
 *
 *  3. Neither the name of the Corporation nor the names of the
 *  contributors may be used to endorse or promote products derived from
 *  this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
 *  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 *  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
 *  CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 *  EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 *  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 *  PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 *  LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 *  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 *************************************************************************
 */

#include <iostream>
#include <iomanip>
#include <sys/time.h>

#include <Kokkos_DeviceHost.hpp>
#include <Kokkos_DeviceHost_ValueView.hpp>
#include <Kokkos_DeviceHost_MultiVectorView.hpp>
#include <Kokkos_DeviceHost_MDArrayView.hpp>
#include <Kokkos_DeviceHost_ParallelFor.hpp>
#include <Kokkos_DeviceHost_ParallelReduce.hpp>

#include <Kokkos_DeviceCuda.hpp>
#include <Kokkos_DeviceCuda_ValueView.hpp>
#include <Kokkos_DeviceCuda_MultiVectorView.hpp>
#include <Kokkos_DeviceCuda_MDArrayView.hpp>
#include <Kokkos_DeviceCuda_ParallelFor.hpp>
#include <Kokkos_DeviceCuda_ParallelReduce.hpp>

#include <Kokkos_DeviceCuda_macros.hpp>
#include <CRSMesh.hpp>
#include <assemble.hpp>
#include <CRSMatrixGatherFill.hpp>
#include <Dirichlet.hpp>
#include <CG_Solve.hpp>
#include <driver.hpp>
#include <Kokkos_DeviceClear_macros.hpp>

__global__ void dummy_kernel(){}

namespace Test {

void test_Cuda(int beg, int end, int runs){

  std::cout << "\"MiniFE with Kokkos Host\"" << std::endl;
  std::cout << "\"Size\" , \"Setup\" , \"Populate\" , \"Solve\"" << std::endl
            << "\"elements\" , \"seconds\" , \"seconds\" , \"MFlop/sec\"" << std::endl ;

  for(int i = beg ; i < end; i+=2)
  {
    const int ix = i ;
    const int iy = ix + 1 ;
    const int iz = iy + 1 ;
    const int n  = ix * iy * iz ;

    double times[3], mins[3];

    for(int j = 0; j < runs; j++){

     run_kernel<Kokkos::DeviceCuda>(ix,iy,iz,times);

     if(j == 0) {
       mins[0] = times[0];
       mins[1] = times[1];
       mins[2] = times[2];
     }

     for(int k = 0 ; k < 3 ; k++)
     {
       if(times[k] < mins[k]) mins[k] = times[k];
     }
   }
   std::cout << n << " , " << mins[0] << " , " << mins[1] << " , " << mins[2] << std::endl ;
  }
}
}// namespace

