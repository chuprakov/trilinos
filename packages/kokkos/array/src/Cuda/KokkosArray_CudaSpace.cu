/*
//@HEADER
// ************************************************************************
// 
//   KokkosArray: Manycore Performance-Portable Multidimensional Arrays
//              Copyright (2012) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact  H. Carter Edwards (hcedwar@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <stdlib.h>
#include <iostream>
#include <sstream>

#include <KokkosArray_CudaSpace.hpp>
#include <Cuda/KokkosArray_Cuda_Internal.hpp>
#include <Cuda/KokkosArray_Cuda_Parallel.hpp>
#include <impl/KokkosArray_MemoryTracking.hpp>
#include <impl/KokkosArray_Error.hpp>

/*--------------------------------------------------------------------------*/

namespace KokkosArray {
namespace {

Impl::MemoryTracking & cuda_space_singleton()
{
  static Impl::MemoryTracking self("KokkosArray::CudaSpace");
  return self ;
}

}

/*--------------------------------------------------------------------------*/

DeepCopy<HostSpace,CudaSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{
  CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) );
}

DeepCopy<CudaSpace,HostSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{
  CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) );
}

DeepCopy<CudaSpace,CudaSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{
  CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) );
}

/*--------------------------------------------------------------------------*/

void * CudaSpace::allocate(
  const std::string    & label ,
  const std::type_info & scalar_type ,
  const size_t           scalar_size ,
  const size_t           scalar_count )
{
  HostSpace::assert_master_thread( "KokkosArray::CudaSpace::allocate" );

  const size_t size = scalar_size * scalar_count ;

  void * ptr = 0 ;

  if ( 0 < scalar_size * scalar_count ) {
    bool ok = true ;

    hipDeviceSynchronize();

    if ( ok ) ok = hipSuccess == hipMalloc( & ptr , size );
    if ( ok ) ok = 0 != ptr ;
    if ( ok ) ok = hipSuccess == hipMemset( ptr , 0 , size );
    if ( ok ) ok = hipSuccess == hipDeviceSynchronize();

    if ( ! ok ) {
      std::ostringstream msg ;
      msg << "KokkosArray::Impl::CudaSpace::allocate( "
          << label
          << " , " << scalar_type.name()
          << " , " << scalar_size
          << " , " << scalar_count
          << " ) FAILED memory allocation" ;
      KokkosArray::Impl::throw_runtime_exception( msg.str() );
    }

    cuda_space_singleton()
      .track( ptr, & scalar_type, scalar_size, scalar_count, label );
  }

  return ptr ;
}

#if ! defined( __CUDA_ARCH__ )

void CudaSpace::increment( const void * ptr )
{
  HostSpace::assert_master_thread( "KokkosArray::CudaSpace::increment" );

  if ( 0 != ptr ) {
    cuda_space_singleton().increment( ptr );
  }
}

void CudaSpace::decrement( const void * ptr )
{
  HostSpace::assert_master_thread( "KokkosArray::CudaSpace::decrement" );

  if ( 0 != ptr ) {

    void * ptr_alloc = cuda_space_singleton().decrement( ptr );

    if ( 0 != ptr_alloc ) {

      hipDeviceSynchronize();

      const bool failed = hipSuccess != hipFree( ptr_alloc );

      if ( failed ) {
        std::string msg("KokkosArray::Impl::CudaSpace::decrement() failed hipFree");
        KokkosArray::Impl::throw_runtime_exception( msg );
      }
    }
  }
}

#endif

void CudaSpace::print_memory_view( std::ostream & o )
{
  cuda_space_singleton().print( o , std::string("  ") );
}


size_t CudaSpace::preferred_alignment(
  size_t scalar_size , size_t scalar_count )
{
  const size_t alignment = Impl::CudaTraits::WarpSize * sizeof(size_type);

  // If the array is larger than the warp-alignment
  // then align the count on the warp boundary.

  if ( alignment < scalar_size * scalar_count &&
       0 == alignment % scalar_size ) {
    const size_t align = alignment / scalar_size ;
    const size_t rem   = scalar_count % align ;
    if ( rem ) scalar_count += align - rem ;
  }
  return scalar_count ;
}

std::string CudaSpace::query_label( const void * p )
{
  const Impl::MemoryTracking::Info info =
    cuda_space_singleton().query( p );

  return info.label ;
}

void CudaSpace::access_error()
{
  const std::string msg("KokkosArray::CudaSpace::access_error attempt to execute Cuda function from non-Cuda space" );

  KokkosArray::Impl::throw_runtime_exception( msg );
}

void CudaSpace::access_error( const void * const ptr )
{
  std::ostringstream msg ;
  msg << "KokkosArray::CudaSpace::access_error:" ;
  msg << " attempt to access Cuda-data labeled(" ;
  msg << query_label( ptr ) ;
  msg << ") from non-Cuda execution" ;
  KokkosArray::Impl::throw_runtime_exception( msg.str() );
}

/*--------------------------------------------------------------------------*/

} // namespace KokkosArray

