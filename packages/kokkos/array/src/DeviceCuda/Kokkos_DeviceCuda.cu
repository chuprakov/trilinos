/** \HEADER
 *************************************************************************
 *
 *                            Kokkos
 *                 Copyright 2010 Sandia Corporation
 *
 *  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
 *  the U.S. Government retains certain rights in this software.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are
 *  met:
 *
 *  1. Redistributions of source code must retain the above copyright
 *  notice, this list of conditions and the following disclaimer.
 *
 *  2. Redistributions in binary form must reproduce the above copyright
 *  notice, this list of conditions and the following disclaimer in the
 *  documentation and/or other materials provided with the distribution.
 *
 *  3. Neither the name of the Corporation nor the names of the
 *  contributors may be used to endorse or promote products derived from
 *  this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
 *  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 *  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
 *  CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 *  EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 *  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 *  PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 *  LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 *  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 *************************************************************************
 */

#include <stdlib.h>
#include <iostream>
#include <stdexcept>
#include <sstream>

#include <Kokkos_DeviceCuda.hpp>
#include <DeviceCuda/Kokkos_DeviceCuda_DeepCopy.hpp>
#include <impl/Kokkos_MemoryInfo.hpp>

/*--------------------------------------------------------------------------*/

namespace Kokkos {

namespace {

void cuda_safe_call( hipError_t e , const char * name )
{
  if ( hipSuccess != e ) {
    std::ostringstream out ;
    out << name << " error: " << hipGetErrorString(e);
    throw std::runtime_error( out.str() );
  }
}

#define CUDA_SAFE_CALL( call )  cuda_safe_call( call , # call )

/*--------------------------------------------------------------------------*/

class DeviceCuda_Impl {
public:
  Impl::MemoryInfoSet            m_allocations ;
  struct hipDeviceProp_t          m_cudaProp ;
  int                            m_cudaDev ;
  unsigned                       m_maxWarp ;
  DeviceCuda::Traits::WordType * m_reduceScratchSpace ;
  DeviceCuda::Traits::WordType * m_reduceScratchFlag ;

  explicit DeviceCuda_Impl( int cuda_device_id );
  ~DeviceCuda_Impl();

  static DeviceCuda_Impl & singleton( int cuda_device_id = 0 );

  void * allocate_memory(
    const std::string    & label ,
    const std::type_info & type ,
    const size_t member_size ,
    const size_t member_count );

  void deallocate_memory( void * ptr );

private:
  DeviceCuda_Impl();
  DeviceCuda_Impl( const DeviceCuda_Impl & );
  DeviceCuda_Impl & operator = ( const DeviceCuda_Impl & );
};

DeviceCuda_Impl & DeviceCuda_Impl::singleton( int cuda_device_id )
{
  static DeviceCuda_Impl self( cuda_device_id );
  return self ;
}

DeviceCuda_Impl::DeviceCuda_Impl( int cuda_device_id )
  : m_allocations()
  , m_cudaProp()
  , m_cudaDev( cuda_device_id )
  , m_maxWarp( 0 )
  , m_reduceScratchSpace( 0 )
  , m_reduceScratchFlag( 0 )
{
  // Some significant cuda device properties:
  //
  // hipDeviceProp_t::major               : Device major number
  // hipDeviceProp_t::minor               : Device minor number
  // hipDeviceProp_t::multiProcessorCount : number of multiprocessors
  // hipDeviceProp_t::sharedMemPerBlock   : capacity of shared memory per block
  // hipDeviceProp_t::totalGlobalMem      : capacity of global memory

  enum { n = sizeof(DeviceCuda::Traits::WordType) };

  const DeviceCuda::Traits::WordType zero = 0 ;

  // Device query

  CUDA_SAFE_CALL( hipGetDevice( & m_cudaDev ) );
  CUDA_SAFE_CALL( hipGetDeviceProperties( & m_cudaProp , m_cudaDev ) );

  // Maximum number of warps,
  // at most one warp per thread in a warp for reduction.

  m_maxWarp = DeviceCuda::Traits::WarpSize ;
  while ( m_cudaProp.maxThreadsPerBlock <
          DeviceCuda::Traits::WarpSize * m_maxWarp ) {
    m_maxWarp >>= 1 ;
  }

  // Allocate shared memory image for multiblock reduction scratch space

  const size_t sharedWord =
   ( m_cudaProp.sharedMemPerBlock + n - 1 ) / n ;

  m_reduceScratchSpace = (DeviceCuda::Traits::WordType *)
    allocate_memory( std::string("MultiblockReduceScratchSpace") ,
                     typeid( DeviceCuda::Traits::WordType ),
                     sizeof( DeviceCuda::Traits::WordType ),
                     sharedWord + 1 );

  m_reduceScratchFlag = m_reduceScratchSpace + sharedWord ;

  CUDA_SAFE_CALL(
    hipMemcpy( m_reduceScratchFlag , & zero , n, hipMemcpyHostToDevice ) );
}

DeviceCuda_Impl::~DeviceCuda_Impl()
{
  deallocate_memory( m_reduceScratchSpace );

  m_reduceScratchSpace = 0 ;
  m_reduceScratchFlag  = 0 ;

  if ( ! m_allocations.empty() ) {
    std::cerr << "Kokkos::DeviceCuda memory leaks:" << std::endl ;
    m_allocations.print( std::cerr );
  }
}

void * DeviceCuda_Impl::allocate_memory(
  const std::string    & label ,
  const std::type_info & type ,
  const size_t member_size ,
  const size_t member_count )
{
  Impl::MemoryInfo tmp ;

  tmp.m_type  = & type ;
  tmp.m_label = label ;
  tmp.m_size  = member_size ;
  tmp.m_count = member_count ;

  CUDA_SAFE_CALL( hipMalloc( & tmp.m_ptr , member_size * member_count ) );

  const bool ok_alloc  = 0 != tmp.m_ptr ;
  const bool ok_insert = ok_alloc && m_allocations.insert( tmp );

  if ( ! ok_alloc || ! ok_insert ) {
    std::ostringstream msg ;
    msg << "Kokkos::DeviceCuda::allocate_memory( " << label
        << " , " << type.name()
        << " , " << member_size
        << " , " << member_count
        << " ) FAILED " ;
    if ( ok_alloc ) { msg << "memory allocation" ; }
    else            { msg << "with internal error" ; }
    throw std::runtime_error( msg.str() );
  }

  return tmp.m_ptr ;
}

void DeviceCuda_Impl::deallocate_memory( void * ptr )
{
  if ( ! m_allocations.erase( ptr ) ) {
    std::ostringstream msg ;
    msg << "Kokkos::DeviceCuda::deallocate_memory( " << ptr
        << " ) FAILED memory allocated by this device" ;
    throw std::runtime_error( msg.str() );
  }

  CUDA_SAFE_CALL( hipFree( ptr ) );
}

}

/*--------------------------------------------------------------------------*/

void DeviceCuda::initialize( int cuda_device_id )
{
  DeviceCuda_Impl::singleton( cuda_device_id );
}

DeviceCuda::Traits::WordType *
DeviceCuda::reduce_multiblock_scratch_space()
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();
  return s.m_reduceScratchSpace ;
}

DeviceCuda::Traits::WordType *
DeviceCuda::reduce_multiblock_scratch_flag()
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();
  return s.m_reduceScratchFlag ;
}

DeviceCuda::size_type
DeviceCuda::maximum_warp_count()
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();
  return s.m_maxWarp ;
}

DeviceCuda::size_type
DeviceCuda::maximum_grid_count()
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();
  return s.m_cudaProp.maxGridSize[0];
}

/*--------------------------------------------------------------------------*/

void * DeviceCuda::allocate_memory(
  const std::string    & label ,
  const std::type_info & type ,
  const size_t member_size ,
  const size_t member_count )
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();

  return s.allocate_memory( label ,type , member_size , member_count );
}

void DeviceCuda::deallocate_memory( void * ptr )
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();

  s.deallocate_memory( ptr );
}

void DeviceCuda::print_memory_view( std::ostream & o )
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();

  s.m_allocations.print( o );
}

/*--------------------------------------------------------------------------*/

unsigned int DeviceCuda::m_launching_kernel = false ;

void DeviceCuda::set_dispatch_functor()
{
  if ( m_launching_kernel ) {
    std::string msg ;
    msg.append( "Kokkos::DeviceCuda::set_dispatch_functor FAILED: " );
    msg.append( "kernel dispatch is already in progress, " );
    msg.append( "a recursive call or forgotten 'clear_dispatch_kernel" );
    throw std::runtime_error( msg );
  }
  m_launching_kernel = true ;
}

void DeviceCuda::clear_dispatch_functor()
{
  if ( ! m_launching_kernel ) {
    std::string msg ;
    msg.append( "Kokkos::DeviceCuda::clear_dispatch_functor FAILED: " );
    msg.append( "no kernel dispatch in progress." );
    throw std::runtime_error( msg );
  }
  m_launching_kernel = false ;
}

} // namespace Kokkos

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

namespace Kokkos {
namespace Impl {

void copy_to_cuda_from_host( void * dst , const void * src ,
                             size_t member_size , size_t member_count )
{
  CUDA_SAFE_CALL(
    hipMemcpy( dst , src , member_size * member_count , hipMemcpyHostToDevice ) );

}

void copy_to_host_from_cuda( void * dst , const void * src ,
                             size_t member_size , size_t member_count )
{
  CUDA_SAFE_CALL(
    hipMemcpy( dst , src , member_size * member_count , hipMemcpyDeviceToHost ) );
}

}
}

