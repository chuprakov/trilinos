/** \HEADER
 *************************************************************************
 *
 *                            Kokkos
 *                 Copyright 2010 Sandia Corporation
 *
 *  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
 *  the U.S. Government retains certain rights in this software.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions are
 *  met:
 *
 *  1. Redistributions of source code must retain the above copyright
 *  notice, this list of conditions and the following disclaimer.
 *
 *  2. Redistributions in binary form must reproduce the above copyright
 *  notice, this list of conditions and the following disclaimer in the
 *  documentation and/or other materials provided with the distribution.
 *
 *  3. Neither the name of the Corporation nor the names of the
 *  contributors may be used to endorse or promote products derived from
 *  this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
 *  EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 *  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 *  PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
 *  CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 *  EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 *  PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 *  PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
 *  LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 *  NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 *  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 *************************************************************************
 */

#include <stdlib.h>
#include <iostream>
#include <stdexcept>
#include <sstream>

#include <Kokkos_DeviceCuda.hpp>
#include <impl/Kokkos_MemoryInfo.hpp>

/*--------------------------------------------------------------------------*/

namespace Kokkos {

namespace {

void cuda_safe_call( hipError_t e , const char * name )
{
  if ( hipSuccess != e ) {
    std::ostringstream out ;
    out << name << " error: " << hipGetErrorString(e);
    throw std::runtime_error( out.str() );
  }
}

#define CUDA_SAFE_CALL( call )  cuda_safe_call( call , # call )

/*--------------------------------------------------------------------------*/

class DeviceCuda_Impl {
public:
  Impl::MemoryInfoSet            m_allocations ;
  struct hipDeviceProp_t          m_cudaProp ;
  int                            m_cudaDev ;
  CudaDevice::Traits::WordType * m_reduceScratchSpace ;
  CudaDevice::Traits::WordType * m_reduceScratchFlag ;

  explicit DeviceCuda_Impl( int cuda_device_id );
  ~DeviceCuda_Impl();

  static DeviceCuda_Impl & singleton( int cuda_device_id = 0 );

private:
  DeviceCuda_Impl();
  DeviceCuda_Impl( const DeviceCuda_Impl & );
  DeviceCuda_Impl & operator = ( const DeviceCuda_Impl & );
};

DeviceCuda_Impl & DeviceCuda_Impl::singleton( int cuda_device_id )
{
  static DeviceCuda_Impl self( cuda_device_id );
  return self ;
}

DeviceCuda_Impl::DeviceCuda_Impl( cuda_device_id )
  : m_allocations()
  , m_cudaProp()
  , m_cudaDev( cuda_device_id )
  , m_reduceScratchSpace( 0 )
  , m_reduceScratchFlag( 0 )
{
  // Some significant cuda device properties:
  //
  // hipDeviceProp_t::major               : Device major number
  // hipDeviceProp_t::minor               : Device minor number
  // hipDeviceProp_t::multiProcessorCount : number of multiprocessors
  // hipDeviceProp_t::sharedMemPerBlock   : capacity of shared memory per block
  // hipDeviceProp_t::totalGlobalMem      : capacity of global memory

  enum { n = sizeof(DeviceCuda::Traits::WordType) };

  const DeviceCuda::Traits::WordType zero = 0 ;

  // Device query

  CUDA_SAFE_CALL( hipGetDevice( & m_cudaDev ) );
  CUDA_SAFE_CALL( hipGetDeviceProperties( & m_cudaProp , m_cudaDev ) );

  // Allocate shared memory image for multiblock reduction scratch space

  const size_t sharedWord =
   ( m_cudaProp.sharedMemPerBlock + n - 1 ) / n ;

  m_reduceScratchSpace =
    allocate_memory( std::string("MultiblockReduceScratchSpace") ,
                     typeid( DeviceCuda::Traits::WordType ),
                     sizeof( DeviceCuda::Traits::WordType ),
                     sharedWord + 1 );

  m_reduceScratchFlag = m_reduceScratchSpace + sharedWord ;

  CUDA_SAFE_CALL(
    hipMemcpy( m_reduceScratchFlag , & zero , n, hipMemcpyHostToDevice ) );
}

DeviceCuda_Impl::~DeviceCuda_Impl()
{
  deallocate_memory( m_reduceScratchSpace );

  m_reduceScratchSpace = 0 ;
  m_reduceScratchFlag  = 0 ;

  if ( ! m_allocations.empty() ) {
    std::cerr << "Kokkos::DeviceCuda memory leaks:" << std::endl ;
    m_allocations.print( std::cerr );
  }
}

}

/*--------------------------------------------------------------------------*/

DeviceCuda::initialize( int cuda_device_id )
{
  DeviceCuda_Impl::singleton( cuda_device_id );
}

DeviceCuda::Traits::WordType *
DeviceCuda::reduce_multiblock_scratch_space()
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();
  return s.m_reduceScratchSpace ;
}

DeviceCuda::Traits::WordType *
DeviceCuda::reduce_multiblock_scratch_flag()
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();
  return s.m_reduceScratchFlag ;
}

/*--------------------------------------------------------------------------*/

void * DeviceCuda::allocate_memory(
  const std::string    & label ,
  const std::type_info & type ,
  const size_t member_size ,
  const size_t member_count )
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();

  Impl::MemoryInfo tmp ;

  tmp.m_type  = & type ;
  tmp.m_label = label ;
  tmp.m_size  = member_size ;
  tmp.m_count = member_count ;
  tmp.m_ptr   = calloc( member_size , member_count );

  const bool ok_alloc  = 0 != tmp.m_ptr ;
  const bool ok_insert = ok_alloc && s.m_allocations.insert( tmp );

  if ( ! ok_alloc || ! ok_insert ) {
    std::ostringstream msg ;
    msg << "Kokkos::DeviceCuda::allocate_memory( " << label
        << " , " << type.name()
        << " , " << member_size
        << " , " << member_count
        << " ) FAILED " ;
    if ( ok_alloc ) { msg << "memory allocation" ; }
    else            { msg << "with internal error" ; }
    throw std::runtime_error( msg.str() );
  }

  return tmp.m_ptr ;
}

void DeviceCuda::deallocate_memory( void * ptr )
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();

  if ( ! s.m_allocations.erase( ptr ) ) {
    std::ostringstream msg ;
    msg << "Kokkos::DeviceCuda::deallocate_memory( " << ptr
        << " ) FAILED memory allocated by this device" ;
    throw std::runtime_error( msg.str() );
  }

  free( ptr );
}

void DeviceCuda::print_memory_view( std::ostream & o )
{
  DeviceCuda_Impl & s = DeviceCuda_Impl::singleton();

  s.m_allocations.print( o );
}

/*--------------------------------------------------------------------------*/

unsigned int DeviceCuda::m_launching_kernel = false ;

void DeviceCuda::set_dispatch_functor()
{
  if ( m_launching_kernel ) {
    std::string msg ;
    msg.append( "Kokkos::DeviceCuda::set_dispatch_functor FAILED: " );
    msg.append( "kernel dispatch is already in progress, " );
    msg.append( "a recursive call or forgotten 'clear_dispatch_kernel" );
    throw std::runtime_error( msg );
  }
  m_launching_kernel = true ;
}

void DeviceCuda::clear_dispatch_functor()
{
  if ( ! m_launching_kernel ) {
    std::string msg ;
    msg.append( "Kokkos::DeviceCuda::clear_dispatch_functor FAILED: " );
    msg.append( "no kernel dispatch in progress." );
    throw std::runtime_error( msg );
  }
  m_launching_kernel = false ;
}


} // namespace Kokkos

