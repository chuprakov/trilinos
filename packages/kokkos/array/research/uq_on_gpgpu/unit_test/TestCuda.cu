/*
//@HEADER
// ************************************************************************
// 
//   KokkosArray: Manycore Performance-Portable Multidimensional Arrays
//              Copyright (2012) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact  H. Carter Edwards (hcedwar@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <KokkosArray_Host.hpp>
#include <KokkosArray_Cuda.hpp>

#include <KokkosArray_ProductTensor.hpp>
#include <KokkosArray_LegendrePolynomial.hpp>
#include <KokkosArray_SymmetricDiagonalSpec.hpp>
#include <KokkosArray_StochasticProductTensor.hpp>
#include <KokkosArray_BlockCrsMatrix.hpp>
#include <KokkosArray_CrsMatrix.hpp>

//




#include <Host/KokkosArray_Host_ProductTensor.hpp>

#include <Cuda/KokkosArray_Cuda_SymmetricDiagonalSpec.hpp>
#include <Cuda/KokkosArray_Cuda_ProductTensor.hpp>
#include <Cuda/KokkosArray_Cuda_CrsProductTensorLegendre.hpp>
#include <Cuda/KokkosArray_Cuda_StochasticProductTensor.hpp>
#include <Cuda/KokkosArray_Cuda_BlockCrsMatrix.hpp>
#include <Cuda/KokkosArray_Cuda_CrsMatrix.hpp>

//

#include <TestBlockCrsMatrix.hpp>
#include <TestTensorCrsMatrix.hpp>
#include <TestStochastic.hpp>

namespace unit_test {

template<>
void performance_test_driver<KokkosArray::Cuda>(bool test_flat, bool test_orig, bool test_block, bool check)
{
  typedef KokkosArray::Cuda Device;

  int nGrid;
  int nIter; 
  bool print;

  

  // All methods compared against flat-original
  if (test_flat) {
    nGrid = 5 ;
    nIter = 1 ; 
    print = false ;
    performance_test_driver_all<Device>( 3 , 1 ,  9 , nGrid , nIter , print ,
    					 test_block , check );
    performance_test_driver_all<Device>( 5 , 1 ,  5 , nGrid , nIter , print ,
					 test_block , check );
  }

#ifdef HAVE_KOKKOSARRAY_STOKHOS
  // Just polynomial methods compared against original
  if (test_orig) {
    nGrid = 32 ;
    nIter = 1 ; 
    print = false ;
    performance_test_driver_poly<Device>( 3 , 1 , 12 , nGrid , nIter , print , 
    					  test_block , check );
    performance_test_driver_poly<Device>( 5 , 1 ,  6 , nGrid , nIter , print ,
					  test_block , check );
  }
#endif

  //------------------------------

  /*
  std::cout << std::endl
            << "\"CRS flat-matrix ~27 nonzeros/row (CUDA uses cusparse)\""
            << std::endl
	    << "\"nGrid\" , "
            << "\"VectorSize\" , "
            << "\"MXV-Time\""
            << std::endl ;

  for ( int n_grid = 10 ; n_grid <= 100 ; n_grid += 5 ) {

    const std::pair<size_t,double> perf_flat =
      test_flat_matrix<double,Device>( n_grid , nIter , print );

    std::cout << n_grid << " , "
	      << perf_flat.first << " , "
              << perf_flat.second
              << std::endl ;
  }
  */

  //------------------------------
}

}

int mainCuda(bool test_flat, bool test_orig, bool test_block, bool check, 
	     int device_id)
{
  typedef unsigned long long int IntType ;

  KokkosArray::Cuda::initialize( KokkosArray::Cuda::SelectDevice(0) );

  hipSetDevice(device_id);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  std::cout << std::endl 
	    << "Device " << device_id << ": " << deviceProp.name 
	    << std::endl;

//  unit_test::test_dense<KokkosArray::Cuda>();
//  unit_test::test_diagonal<KokkosArray::Cuda>();
//  unit_test::test_other<KokkosArray::Cuda>();

//  unit_test::test_inner_product_legengre_polynomial<10,KokkosArray::Cuda>();
//  unit_test::test_triple_product_legendre_polynomial<4,KokkosArray::Cuda>();

  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 1 , 2 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 1 , 5 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 2 , 1 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 2 , 2 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 3 , 1 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 3 , 2 );

  unit_test_tensor::test_tensor_crs_matrix<KokkosArray::Cuda,IntType>( 1 , 2 );
  unit_test_tensor::test_tensor_crs_matrix<KokkosArray::Cuda,IntType>( 1 , 5 );
  unit_test_tensor::test_tensor_crs_matrix<KokkosArray::Cuda,IntType>( 2 , 1 );
  unit_test_tensor::test_tensor_crs_matrix<KokkosArray::Cuda,IntType>( 5 , 1 );
  unit_test_tensor::test_tensor_crs_matrix<KokkosArray::Cuda,IntType>( 5 , 5 );

  std::cout << "Stress tests:" << std::endl ;

  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 10 , 8 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 11 , 8 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 12 , 10 );
  unit_test::test_block_crs_matrix<KokkosArray::Cuda>( 13 , 10 );

  unit_test_tensor::test_tensor_crs_matrix<KokkosArray::Cuda,IntType>( 100 , 10 );

  std::cout << std::endl << "\"Cuda Performance\"" << std::endl ;
  unit_test::performance_test_driver<KokkosArray::Cuda>(
    test_flat, test_orig, test_block, check);

  KokkosArray::Cuda::finalize();

  return 0 ;
}

