#include "hip/hip_runtime.h"
/*
//@HEADER
// ************************************************************************
// 
//          Kokkos: Node API and Parallel Node Kernels
//              Copyright (2008) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact Michael A. Heroux (maherou@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <Kokkos_DeviceCuda.hpp>
#include <Kokkos_DeviceCuda_ValueView.hpp>
#include <Kokkos_DeviceCuda_MultiVectorView.hpp>
#include <Kokkos_DeviceCuda_MDArrayView.hpp>
#include <Kokkos_DeviceCuda_ParallelFor.hpp>
#include <Kokkos_DeviceCuda_ParallelReduce.hpp>

#include <Kokkos_DeviceCuda_macros.hpp>
#include <sstream>
#include "test_run.hpp"

__global__ void dummy_kernel(){}

int main (int argc, char* argv[]) {
  using std::cerr;
  using std::endl;

  int numInnerLoops = 1000;
  int numOuterLoops = 10;
  if (argc > 1) {
    std::istringstream is (argv[1]);
    is >> numInnerLoops;
    if (! is) {
      cerr << "Failed to read number of inner loops (first argument)" << endl;
      return -1;
    }
  }
  if (argc > 2) {
    std::istringstream is (argv[2]);
    is >> numOuterLoops;
    if (! is) {
      cerr << "Failed to read number of outer loops (second argument)" << endl;
      return -2;
    }
  }

  test_run<double, Kokkos::DeviceCuda> ("DeviceCuda", numInnerLoops, numOuterLoops);
  return 0;
}
