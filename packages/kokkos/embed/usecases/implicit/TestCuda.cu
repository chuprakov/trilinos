
#include <Kokkos_Cuda.hpp>

#include <ParallelComm.hpp>
#include <TestImplicit.hpp>

namespace Test {

int test_cuda( comm::Machine machine , std::istream & input )
{
  const unsigned parallel_rank = comm::rank( machine );
  const unsigned device_count  = Kokkos::Cuda::detect_device_count();

  unsigned device = 0 ;
  unsigned elem_beg = 3 ;
  unsigned elem_end = 4 ;
  unsigned run = 1 ;

  while ( ! input.eof() ) {
    std::string which ;

    input >> which ;

    if ( which == std::string("device") ) {
      input >> device ;
    }
    else if ( which == std::string("implicit") ) {
      input >> elem_beg ;
      input >> elem_end ;
      input >> run ;
    }
    else {
      std::cerr << "Expected \"device #Device\" OR \"implicit #ElemBeg #ElemEnd #Run\""
                << std::endl ;
      return -1 ;
    }
  }

  device += parallel_rank % device_count ;

  Kokkos::Cuda::initialize( Kokkos::Cuda::SelectDevice( device ) );

  {
    std::ostringstream label ;

    label << "Scalar, CudaArch[" << Kokkos::Cuda::detect_device_arch()[device] << "]" ;

    implicit_driver<double,Kokkos::Cuda>(
      label.str().c_str() , machine , 1 , elem_beg , elem_end , run );
  }

  {
    std::ostringstream label ;

    label << "Ensemble[32], CudaArch[" << Kokkos::Cuda::detect_device_arch()[device] << "]" ;

    implicit_driver< Kokkos::Array<double,32> , Kokkos::Cuda>(
      label.str().c_str() , machine , 1 , elem_beg , elem_end , run );
  }

  Kokkos::Cuda::finalize();

  return 0 ;
}

}

