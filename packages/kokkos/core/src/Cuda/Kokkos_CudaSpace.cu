/*
//@HEADER
// ************************************************************************
// 
//   Kokkos: Manycore Performance-Portable Multidimensional Arrays
//              Copyright (2012) Sandia Corporation
// 
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions? Contact  H. Carter Edwards (hcedwar@sandia.gov) 
// 
// ************************************************************************
//@HEADER
*/

#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <stdexcept>

#include <Kokkos_Cuda.hpp>
#include <Kokkos_CudaSpace.hpp>

#include <Cuda/Kokkos_Cuda_Internal.hpp>
#include <impl/Kokkos_MemoryTracking.hpp>
#include <impl/Kokkos_Error.hpp>

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

namespace Kokkos {
namespace Impl {

DeepCopy<CudaSpace,CudaSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{ CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) ); }

DeepCopy<HostSpace,CudaSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{ CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) ); }

DeepCopy<CudaSpace,HostSpace>
  ::DeepCopy( void * dst , const void * src , size_t n )
{ CUDA_SAFE_CALL( hipMemcpy( dst , src , n , hipMemcpyDefault ) ); }

} // namespace Impl
} // namespace Kokkos

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

namespace Kokkos {
namespace Impl {
namespace {

class CudaMemoryTracking {
public:

  enum SpaceTag { CudaSpaceTag , CudaUVMSpaceTag , CudaHostPinnedSpaceTag };

  struct Attribute {

    Kokkos::Impl::cuda_texture_object_type m_tex_obj ;

    Attribute() : m_tex_obj(0) {}

    ~Attribute()
      {
        if ( m_tex_obj ) {
          hipDestroyTextureObject( m_tex_obj );
          m_tex_obj = 0 ;
        }
      }

    hipError_t create( void * const                  arg_alloc_ptr
                    , size_t const                  arg_byte_size
                    , hipChannelFormatDesc const & arg_desc
                    )
    {
      hipError_t cuda_status = hipSuccess ;

      if ( 0 == m_tex_obj ) {
 
        cuda_status = hipDeviceSynchronize();

        struct hipResourceDesc resDesc ;
        struct hipTextureDesc  texDesc ;

        memset( & resDesc , 0 , sizeof(resDesc) );
        memset( & texDesc , 0 , sizeof(texDesc) );

        resDesc.resType                = hipResourceTypeLinear ;
        resDesc.res.linear.desc        = arg_desc ;
        resDesc.res.linear.sizeInBytes = arg_byte_size ;
        resDesc.res.linear.devPtr      = arg_alloc_ptr ;

        cuda_status = hipCreateTextureObject( & m_tex_obj , & resDesc, & texDesc, NULL);

        if ( hipSuccess == cuda_status ) { cuda_status = hipDeviceSynchronize(); }

        if ( ( hipSuccess == cuda_status ) && ( 0 == m_tex_obj ) ) {
          throw std::logic_error(std::string("FAILED assumption that Cuda texture objects are non-zero"));
        }
      }

      return cuda_status ;
    }
  };

  typedef          Kokkos::Impl::MemoryTracking< Attribute >         tracking_type ;
  typedef typename Kokkos::Impl::MemoryTracking< Attribute >::Entry  entry_type ;

private:

  tracking_type   m_tracking ;
  SpaceTag const  m_space_tag ;


  hipError_t cuda_malloc( void ** ptr , size_t byte_size ) const
    {
      hipError_t result = hipSuccess ;

      switch( m_space_tag ) {
      case CudaSpaceTag :
        result = hipMalloc( ptr , byte_size );
        break ;
      case CudaUVMSpaceTag :
#if defined( CUDA_VERSION ) && ( 6000 <= CUDA_VERSION )
        result = hipMallocManaged( ptr, byte_size, hipMemAttachGlobal );
#else
        Kokkos::Impl::throw_runtime_exception( std::string("CUDA VERSION does not support UVM") );
#endif
        break ;
      case CudaHostPinnedSpaceTag :
        result = hipHostAlloc( ptr , byte_size , hipHostMallocDefault );
        break ;
      }

      return result ;
    }

  hipError_t cuda_free( void * ptr ) const
    {
      hipError_t result = hipSuccess ;

      switch( m_space_tag ) {
      case CudaSpaceTag :
      case CudaUVMSpaceTag :
        result = hipFree( ptr );
        break ;
      case CudaHostPinnedSpaceTag :
        result = hipHostFree( ptr );
        break ;
      }
      return result ;
    }

public :

  CudaMemoryTracking( const SpaceTag arg_tag , const char * const arg_label )
    : m_tracking(  arg_label )
    , m_space_tag( arg_tag )
    {}

  void print( std::ostream & oss , const std::string & lead ) const
    { m_tracking.print( oss , lead ); }

  const char * query_label( const void * ptr ) const
    {
      static const char error[] = "<NOT FOUND>" ;
      entry_type * const entry = m_tracking.query( ptr );
      return entry ? entry->label() : error ;
    }

  void * allocate(
    const std::string    & label ,
    const std::type_info & scalar_type ,
    const size_t           scalar_size ,
    const size_t           scalar_count )
  {
    void * ptr = 0 ;

    const size_t byte_size = scalar_size * scalar_count ;

    if ( byte_size ) {

      const bool ok_parallel = ! HostSpace::in_parallel();

      hipError_t cuda_status = hipSuccess ;

      if ( ok_parallel ) {

        cuda_status = hipDeviceSynchronize();

        if ( hipSuccess == cuda_status ) { cuda_status = CudaMemoryTracking::cuda_malloc( & ptr , byte_size ); }
        if ( hipSuccess == cuda_status ) { cuda_status = hipDeviceSynchronize(); }
      }

      if ( ok_parallel && ( hipSuccess == cuda_status ) ) {
        m_tracking.insert( label , ptr , scalar_size , scalar_count );
      }
      else {
        std::ostringstream msg ;
        msg << m_tracking.label()
            << "::allocate( "
            << label
            << " , " << scalar_type.name()
            << " , " << scalar_size
            << " , " << scalar_count
            << " ) FAILURE : " ;
        if ( ! ok_parallel ) {
          msg << "called within a parallel functor" ;
        }
        else {
          msg << " CUDA ERROR \"" << hipGetErrorString(cuda_status) << "\"" ;
        }
        Kokkos::Impl::throw_runtime_exception( msg.str() );
      }
    }

    return ptr ;
  }

  void decrement( const void * ptr )
  {
    const bool ok_parallel = ! HostSpace::in_parallel();

    hipError_t cuda_status = hipSuccess ;

    if ( ok_parallel ) {

      cuda_status = hipDeviceSynchronize();

      void * const alloc_ptr = ( hipSuccess == cuda_status ) ? m_tracking.decrement( ptr ) : (void *) 0 ;

      if ( alloc_ptr ) {
        if ( hipSuccess == cuda_status ) { cuda_status = CudaMemoryTracking::cuda_free( alloc_ptr ); }
        if ( hipSuccess == cuda_status ) { cuda_status = hipDeviceSynchronize(); }
      }
    }

    if ( ( ! ok_parallel ) || ( hipSuccess != cuda_status ) ) {
      std::ostringstream msg ;
      msg << m_tracking.label() << "::decrement( " << ptr << " ) FAILURE : " ;
      if ( ! ok_parallel ) {
        msg << "called within a parallel functor" ;
      }
      else {
        msg << " CUDA ERROR \"" << hipGetErrorString(cuda_status) << "\"" ;
      }
      std::cerr << msg.str() << std::endl ;
    }
  }

  void increment( const void * ptr )
    {
      const bool ok_parallel = ! HostSpace::in_parallel();

      if ( ok_parallel ) {
        m_tracking.increment( ptr );
      }
      else {
        std::ostringstream msg ;
        msg << m_tracking.label() << "::increment(" << ptr
            << ") FAILURE :called within a parallel functor" ;
        Kokkos::Impl::throw_runtime_exception( msg.str() );
      }
    }


  inline
  void texture_object_attach( const void * const            arg_ptr
                            , const hipChannelFormatDesc & arg_desc
                            , ::hipTextureObject_t * const arg_tex_obj
                            , void const           ** const arg_alloc_ptr
                            , int                   * const arg_offset
                            )
    {
      static const size_t max_array_len = 1 << 28 ;

      *arg_tex_obj   = 0 ;
      *arg_alloc_ptr = 0 ;
      *arg_offset    = 0 ;

      if ( arg_ptr ) {

        // Can only create texture object on device architure 3.0 or better
        const bool ok_dev_arch = 300 <= Cuda::device_arch();
        const bool ok_parallel = ok_dev_arch && ! HostSpace::in_parallel();

        entry_type * const entry = ok_parallel ? m_tracking.query( arg_ptr ) : (entry_type *) 0 ;

        const size_t offset = entry ? ( reinterpret_cast<const char*>(arg_ptr) -
                                        reinterpret_cast<const char*>(entry->m_alloc_ptr) ) : 0 ;

        const bool ok_offset = entry     && ( 0 == ( offset % entry->m_type_size ) );
        const bool ok_count  = ok_offset && ( entry->m_array_len < max_array_len );

        hipError_t cuda_status = hipSuccess ;

        if ( ok_count ) {
          cuda_status = entry->m_attribute.create( entry->m_alloc_ptr
                                                 , entry->m_type_size * entry->m_array_len
                                                 , arg_desc );
        }

        if ( ok_count && ( hipSuccess == cuda_status ) ) {
          *arg_tex_obj   = entry->m_attribute.m_tex_obj ;
          *arg_alloc_ptr = entry->m_alloc_ptr ;
          *arg_offset    = offset / entry->m_type_size ;
        }
        else {
          std::ostringstream msg ;
          msg << m_tracking.label()
              << "::cuda_texture_object_attach(" << arg_ptr << ") FAILED :" ;
          if ( ! ok_dev_arch ) {
            msg << " cuda architecture " << Cuda::device_arch()
                << " does not support texture objects" ;
          }
          else if ( ! ok_parallel ) {
            msg << " called within a parallel functor" ;
          }
          else if ( 0 == entry ) {
            msg << " pointer not tracked" ;
          }
          else if ( ! ok_offset ) {
            msg << " pointer not properly aligned" ;
          }
          else if ( ! ok_count ) {
            msg << " array too large for texture object" ;
          }
          else {
            msg << " CUDA ERROR \"" << hipGetErrorString(cuda_status) << "\"" ;
          }
          Kokkos::Impl::throw_runtime_exception( msg.str() );
        }
      }
    }
};

//----------------------------------------------------------------------------

CudaMemoryTracking &
cuda_space_singleton()
{
#if defined( KOKKOS_USE_CUDA_UVM )
  static CudaMemoryTracking s( CudaMemoryTracking::CudaUVMSpaceTag , "Kokkos::CudaSpace");
#else
  static CudaMemoryTracking s( CudaMemoryTracking::CudaSpaceTag , "Kokkos::CudaSpace");
#endif
  return s ;
}

CudaMemoryTracking &
cuda_uvm_space_singleton()
{
  static CudaMemoryTracking s( CudaMemoryTracking::CudaUVMSpaceTag , "Kokkos::CudaUVMSpace");
  return s ;
}

CudaMemoryTracking &
cuda_host_pinned_space_singleton()
{
  static CudaMemoryTracking s( CudaMemoryTracking::CudaHostPinnedSpaceTag , "Kokkos::CudaHostPinnedSpace");
  return s ;
}

}
} // namespace Impl
} // namespace Kokkos

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

namespace Kokkos {

void * CudaSpace::allocate(
  const std::string    & label ,
  const std::type_info & scalar_type ,
  const size_t           scalar_size ,
  const size_t           scalar_count )
{
  return Impl::cuda_space_singleton().allocate( label , scalar_type , scalar_size , scalar_count );
}

void CudaSpace::decrement( const void * ptr )
{
  Impl::cuda_space_singleton().decrement( ptr );
}


void CudaSpace::increment( const void * ptr )
{
  Impl::cuda_space_singleton().increment( ptr );
}

void CudaSpace::print_memory_view( std::ostream & oss )
{
  Impl::cuda_space_singleton().print( oss , std::string("  ") );
}

std::string CudaSpace::query_label( const void * p )
{
  return std::string( Impl::cuda_space_singleton().query_label(p) );
}

void CudaSpace::texture_object_attach( const void * const            arg_ptr
                                     , ::hipChannelFormatDesc const & arg_desc
                                     , ::hipTextureObject_t * const arg_tex_obj
                                     , void const           ** const arg_alloc_ptr
                                     , int                   * const arg_offset
                                     )
{
  Impl::cuda_space_singleton().texture_object_attach( arg_ptr , arg_desc , arg_tex_obj , arg_alloc_ptr , arg_offset );
}

void CudaSpace::access_error()
{
  const std::string msg("Kokkos::CudaSpace::access_error attempt to execute Cuda function from non-Cuda space" );

  Kokkos::Impl::throw_runtime_exception( msg );
}

void CudaSpace::access_error( const void * const ptr )
{
  std::ostringstream msg ;
  msg << "Kokkos::CudaSpace::access_error:" ;
  msg << " attempt to access Cuda-data labeled(" ;
  msg << query_label( ptr ) ;
  msg << ") from non-Cuda execution" ;
  Kokkos::Impl::throw_runtime_exception( msg.str() );
}

} // namespace Kokkos

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

namespace Kokkos {

void * CudaUVMSpace::allocate(
  const std::string    & label ,
  const std::type_info & scalar_type ,
  const size_t           scalar_size ,
  const size_t           scalar_count )
{
  return Impl::cuda_uvm_space_singleton().allocate( label , scalar_type , scalar_size , scalar_count );
}

void CudaUVMSpace::decrement( const void * ptr )
{
  Impl::cuda_uvm_space_singleton().decrement( ptr );
}


void CudaUVMSpace::increment( const void * ptr )
{
  Impl::cuda_uvm_space_singleton().increment( ptr );
}

void CudaUVMSpace::print_memory_view( std::ostream & oss )
{
  Impl::cuda_uvm_space_singleton().print( oss , std::string("  ") );
}

std::string CudaUVMSpace::query_label( const void * p )
{
  return std::string( Impl::cuda_uvm_space_singleton().query_label(p) );
}

void CudaUVMSpace::texture_object_attach( const void * const            arg_ptr
                                        , ::hipChannelFormatDesc const & arg_desc
                                        , ::hipTextureObject_t * const arg_tex_obj
                                        , void const           ** const arg_alloc_ptr
                                        , int                   * const arg_offset
                                        )
{
  Impl::cuda_uvm_space_singleton().texture_object_attach( arg_ptr , arg_desc , arg_tex_obj , arg_alloc_ptr , arg_offset );
}

} // namespace Kokkos

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

namespace Kokkos {

void * CudaHostPinnedSpace::allocate(
  const std::string    & label ,
  const std::type_info & scalar_type ,
  const size_t           scalar_size ,
  const size_t           scalar_count )
{
  return Impl::cuda_host_pinned_space_singleton().allocate( label , scalar_type , scalar_size , scalar_count );
}

void CudaHostPinnedSpace::decrement( const void * ptr )
{
  Impl::cuda_host_pinned_space_singleton().decrement( ptr );
}


void CudaHostPinnedSpace::increment( const void * ptr )
{
  Impl::cuda_host_pinned_space_singleton().increment( ptr );
}

void CudaHostPinnedSpace::print_memory_view( std::ostream & oss )
{
  Impl::cuda_host_pinned_space_singleton().print( oss , std::string("  ") );
}

std::string CudaHostPinnedSpace::query_label( const void * p )
{
  return std::string( Impl::cuda_host_pinned_space_singleton().query_label(p) );
}

} // namespace Kokkos

/*--------------------------------------------------------------------------*/
/*--------------------------------------------------------------------------*/

